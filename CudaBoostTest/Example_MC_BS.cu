#include "hip/hip_runtime.h"
#pragma warning (disable : 4267)
#pragma warning (disable : 4244)

#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/random/normal_distribution.h>

#include <iostream>
#include <iomanip>
#include <cmath>

#include "Example_MC_BS.cuh"


__host__ __device__
unsigned int hashBS(unsigned int a)
{
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

struct estimate_BS : public thrust::unary_function<unsigned int,float>
{
  __device__
  float operator()(unsigned int thread_id)
  {
    float sum = 0;
    unsigned int N = 100000; // samples per thread

    unsigned int seed = thread_id;

    // seed a random number generator
    thrust::default_random_engine rng(seed);

    // create a mapping from random numbers to N(0,1)
    thrust::random::normal_distribution<float> ndist(0.0f, 1.0f);

    float S0 = 20.0f;
    float sig = 0.28f;
    float r = 0.045f;
    float K = 21.0f;
    float T = 0.5f;

    float sqrtT = sqrtf(T);
    float sig2 = sig*sig;

    // take N samples in a quarter circle
    for(unsigned int i = 0; i < N; ++i)
    {
      float W = ndist(rng);
      float ST =    S0 * expf((r - 0.5f*sig2)*T + sig*sqrtT*W);
      float ST_at = S0 * expf((r - 0.5f*sig2)*T - sig*sqrtT*W);
      sum += (((ST-K > 0.0f)? ST-K:0.0f) + ((ST_at-K > 0.0f)? ST_at-K:0.0f))/2.0f;
    }

    // discount back
    sum *= expf(-r*T);

    // divide by N
    return sum / N;
  }
};

void exmpl_thrust_MC_BS()
{
  // use 30K independent seeds
  int M = 50000;

  float estimate = thrust::transform_reduce(thrust::counting_iterator<int>(0),
                                            thrust::counting_iterator<int>(M),
                                            estimate_BS(),
                                            0.0f,
                                            thrust::plus<float>());
  estimate /= M;

  std::cout << std::setprecision(10);
  std::cout << "Option price is approximately " << estimate << std::endl;
  hipDeviceReset();
};