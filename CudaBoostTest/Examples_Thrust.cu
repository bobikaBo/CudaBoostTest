#include "hip/hip_runtime.h"
#pragma warning (disable : 4267)
#pragma warning (disable : 4244)

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/count.h>
#include <thrust/inner_product.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/iterator/zip_iterator.h>

#include <tuple>
#include <iostream>
#include "Examples_Thrust.h"


struct saxpy_functor
{
  const float a;

  saxpy_functor(float _a) : a(_a) {}

  __host__ __device__
  float operator()(const float& x, const float& y) const
  { 
    return a * x + y;
  }
};

void saxpy_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
  // Y <- A * X + Y
  thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(A));
}

void saxpy_slow(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
{
  thrust::device_vector<float> temp(X.size());
   
  // temp <- A
  thrust::fill(temp.begin(), temp.end(), A);
    
  // temp <- A * X
  thrust::transform(X.begin(), X.end(), temp.begin(), temp.begin(), thrust::multiplies<float>());

  // Y <- A * X + Y
  thrust::transform(temp.begin(), temp.end(), Y.begin(), Y.begin(), thrust::plus<float>());
}

// transformations example
void exmpl_thrust_transformations() {
    
    std::cout << std::endl << "In this example we show the transformation functionality of thrust:" << std::endl;

    // allocate three device_vectors with 10 elements
    thrust::device_vector<int> X(10);
    thrust::device_vector<int> Y(10);
    thrust::device_vector<int> Z(10);

    // initialize X to 0,1,2,3, ....
    thrust::sequence(X.begin(), X.end());

    // compute Y = -X
    thrust::transform(X.begin(), X.end(), Y.begin(), thrust::negate<int>());

    // fill Z with twos
    thrust::fill(Z.begin(), Z.end(), 2);

    // compute Y = X mod 2
    thrust::transform(X.begin(), X.end(), Z.begin(), Y.begin(), thrust::modulus<int>());

    // replace all the ones in Y with tens
    thrust::replace(Y.begin(), Y.end(), 1, 10);

    // print Y
    thrust::copy(Y.begin(), Y.end(), std::ostream_iterator<int>(std::cout, " "));
    std::cout << std::endl;

    // saxpy test
    thrust::device_vector<float> A(10, 1.0);
    thrust::device_vector<float> B(10, 2.0);
    thrust::device_vector<float> C(10, 2.0);
    saxpy_fast(5.0, A, B);
    saxpy_slow(5.0, A, C);
    thrust::copy(B.begin(), B.end(), std::ostream_iterator<float>(std::cout, " "));
    std::cout << std::endl;
    thrust::copy(C.begin(), C.end(), std::ostream_iterator<float>(std::cout, " "));
    std::cout << std::endl;
};


void exmpl_thrust_reduce() {

    std::cout << std::endl << "In this example we show reduce functionality of thrust:" << std::endl;

    // put three 1s in a device_vector
    thrust::device_vector<int> vec(5,0);
    vec[1] = 1;
    vec[3] = 2;
    vec[4] = 1;

    // count the 1s
    int result_count = thrust::count(vec.begin(), vec.end(), 1);
    int result_max =   *thrust::max_element(vec.begin(), vec.end());
    int result_min =   *thrust::min_element(vec.begin(), vec.end());
    int result_innerProduct = thrust::inner_product(vec.begin(), vec.end(),
                                                    vec.begin(), 
                                                    0);
    bool result_isSorted = thrust::is_sorted(vec.begin(), vec.end());

    std::cout << "Input vector: ";
    thrust::copy(vec.begin(), vec.end(), std::ostream_iterator<int>(std::cout, " "));
    std::cout << std::endl;
    std::cout << "Number of 1s in this vector: " << result_count << std::endl;
    std::cout << "Max element in this vector: " << result_max << std::endl;
    std::cout << "Min element in this vector: " << result_min << std::endl;
    std::cout << "Inner product of this vector: " << result_innerProduct << std::endl;
    std::cout << "Is this vector sorted?: " << result_isSorted << std::endl;
};


// square<T> computes the square of a number f(x) -> x*x
template <typename T>
struct square
{
  __host__ __device__
  T operator()(const T& x) const
  { 
    return x * x;
  }
};

void exmpl_thrust_transform_reduce() {

    std::cout << std::endl << "In this example we show transform-reduce functionality of thrust:" << std::endl;

    // generate sequence
    thrust::device_vector<float> d_x(20);
    thrust::sequence(d_x.begin(), d_x.end());

    // setup arguments
    square<float>        unary_op;
    thrust::plus<float> binary_op;
    float init = 0;

    // compute norm
    float norm = std::sqrt( thrust::transform_reduce(d_x.begin(), d_x.end(), unary_op, init, binary_op) );

    std::cout << "Input vector: ";
    thrust::copy(d_x.begin(), d_x.end(), std::ostream_iterator<float>(std::cout, " "));
    std::cout << std::endl << "Norm of this vector is: " << norm << std::endl;
};

void exmpl_thrust_scan() {
    
    std::cout << std::endl << "In this example we show scan functionality of thrust:" << std::endl;
    // generate sequence
    thrust::device_vector<float> d_x(10);
    thrust::sequence(d_x.begin(), d_x.end());

    thrust::device_vector<float> d_is(d_x.size());
    thrust::device_vector<float> d_es(d_x.size());

    // perform inclusive scan
    thrust::inclusive_scan(d_x.begin(), d_x.end(), d_is.begin());
    // perform exclusive scan
    thrust::exclusive_scan(d_x.begin(), d_x.end(), d_es.begin());

    std::cout << "Input vector: ";
    thrust::copy(d_x.begin(), d_x.end(), std::ostream_iterator<float>(std::cout, " "));
    std::cout << std::endl;

    std::cout << "Inclusive scan: ";
    thrust::copy(d_is.begin(), d_is.end(), std::ostream_iterator<float>(std::cout, " "));
    std::cout << std::endl;

    std::cout << "Exclusive scan: ";
    thrust::copy(d_es.begin(), d_es.end(), std::ostream_iterator<float>(std::cout, " "));
    std::cout << std::endl;
};



void exmpl_thrust_sort() {
    std::cout << std::endl << "In this example we show sorting functionality of thrust:" << std::endl;    

    const int N = 6;
    int    keys[N] = {  1,   4,   2,   8,   5,   7};
    char values[N] = {'a', 'b', 'c', 'd', 'e', 'f'};

    std::cout << "Key vector: ";
    for (int i = 0; i < N; ++i) {
        std::cout << keys[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Value vector: ";
    for (int i = 0; i < N; ++i) {
        std::cout << values[i] << " ";
    }
    std::cout << std::endl;
    std::cout << "Value vector after sorting: ";

    thrust::sort_by_key(keys, keys + N, values);

    for (int i = 0; i < N; ++i) {
        std::cout << values[i] << " ";
    }
    std::cout << std::endl;
};

void exmpl_thrust_zip_iterator() {
    
    // initialize vectors
    thrust::device_vector<int>  A(3);
    thrust::device_vector<char> B(3);
    A[0] = 10;  A[1] = 20;  A[2] = 30;
    B[0] = 'x'; B[1] = 'y'; B[2] = 'z';    

    // create iterator
    typedef thrust::device_vector<int>::iterator IntIterator;
    typedef thrust::device_vector<char>::iterator CharIterator;
    typedef  thrust::zip_iterator<thrust::tuple<IntIterator, CharIterator> > intCharTupleIterator;

    intCharTupleIterator first = thrust::make_zip_iterator(thrust::make_tuple(A.begin(), B.begin()));
    intCharTupleIterator last  = thrust::make_zip_iterator(thrust::make_tuple(A.end(),   B.end()));

    thrust::maximum< thrust::tuple<int,char> > binary_op;
    thrust::tuple<int,char> init = first[0];
    thrust::tuple<int, char> result = thrust::reduce(first, last, init, binary_op);

    std::cout << "(" << thrust::get<0>(result) << ", " << thrust::get<1>(result) <<")" << std::endl; 
};