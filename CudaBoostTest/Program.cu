#pragma warning (disable : 4267)
#pragma warning (disable : 4244)

#include <iostream> 
#include <vector> 
#include <string>
#include <hip/hip_runtime.h> 
#include <thrust/version.h>


#include "Examples_Thrust.h"
#include "Examples_Cuda.h"
#include "Examples_Boost.h"
 
 
// Main entry into the program 
int main(void) 
{ 
     

    int major = THRUST_MAJOR_VERSION;
    int minor = THRUST_MINOR_VERSION;

    std::cout << "Thrust v" << major << "." << minor << std::endl;
    
    exmpl_cuda_cube();
    exmpl_thrust_transformations();
    exmpl_thrust_reduce();
    exmpl_thrust_transform_reduce();
    exmpl_boost_foreach();

 
    return 0; 
} 