#include "hip/hip_runtime.h"
#pragma warning (disable : 4267)
#pragma warning (disable : 4244)

#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/random/normal_distribution.h>
#include <thrust/device_ptr.h>

#include <hiprand.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <iomanip>
#include <cmath>

#include "Example_MC_BS.cuh"


const unsigned int DEFAULT_RAND_N = 10000000;
const unsigned int DEFAULT_SEED = 1;

struct estimate_BS3 : public thrust::unary_function<float, float>
{
  __host__ __device__
  float operator()(float W)
  {
    float S0 = 20.0f;
    float sig = 0.28f;
    float r = 0.045f;
    float K = 21.0f;
    float T = 0.5f;

    float sqrtT = sqrtf(T);
    float sig2 = sig*sig;

    float ST =    S0 * expf((r - 0.5f*sig2)*T + sig*sqrtT*W);
    float ST_at = S0 * expf((r - 0.5f*sig2)*T - sig*sqrtT*W);
    
    return expf(-r*T)*(((ST-K > 0.0f)? ST-K:0.0f) +
                       ((ST_at-K > 0.0f)? ST_at-K:0.0f))/2.0f;

  }
};

void exmpl_thrust_MC_BS2()
{
  unsigned int M = 200;
  unsigned int rand_n = DEFAULT_RAND_N;
  unsigned int seed = DEFAULT_SEED;
  hiprandGenerator_t prngGPU;
  hiprandCreateGenerator(&prngGPU, HIPRAND_RNG_PSEUDO_MTGP32);
  hiprandSetPseudoRandomGeneratorSeed(prngGPU, seed);

  float estimate = 0.0f;
  float *d_rand;
  hipMalloc((void **)&d_rand, rand_n * sizeof(float));
  thrust::device_ptr<float> d_rand_b = thrust::device_pointer_cast(d_rand);
  thrust::device_ptr<float> d_rand_e = d_rand_b + rand_n;

  for (unsigned int i = 0; i < M; ++i)
  {
      hiprandGenerateNormal(prngGPU, (float *) d_rand, rand_n, 0.0f, 1.0f);

      estimate += thrust::transform_reduce( d_rand_b,
                                            d_rand_e,
                                            estimate_BS3(),
                                            0.0f,
                                            thrust::plus<float>());
  }

  estimate /= (rand_n*M);

  std::cout << std::setprecision(10);
  std::cout << "Option price is approximately " << estimate << std::endl;
  
  hiprandDestroyGenerator(prngGPU);
  hipFree(d_rand);
  hipDeviceReset();
};