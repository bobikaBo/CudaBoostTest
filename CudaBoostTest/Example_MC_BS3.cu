#include "hip/hip_runtime.h"
#pragma warning (disable : 4267)
#pragma warning (disable : 4244)

#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/random/normal_distribution.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <iomanip>
#include <cmath>

#include "Example_MC_BS.cuh"


struct estimate_BS3 : public thrust::unary_function<unsigned int,float>
{
  __device__
  float operator()(unsigned int thread_id)
  {
    float sum = 0;
    unsigned int N = 100000; // samples per thread

    unsigned int seed = thread_id;

    hiprandState s;
    hiprand_init(seed, 0, 0, &s);

    float S0 = 20.0f;
    float sig = 0.28f;
    float r = 0.045f;
    float K = 21.0f;
    float T = 0.5f;

    float sqrtT = sqrtf(T);
    float sig2 = sig*sig;

    // take N samples in a quarter circle
    for(unsigned int i = 0; i < N; ++i)
    {
      float W = hiprand_normal(&s);
      float ST =    S0 * expf((r - 0.5f*sig2)*T + sig*sqrtT*W);
      float ST_at = S0 * expf((r - 0.5f*sig2)*T - sig*sqrtT*W);
      sum += (((ST-K > 0.0f)? ST-K:0.0f) + ((ST_at-K > 0.0f)? ST_at-K:0.0f))/2.0f;
    }

    // discount back
    sum *= expf(-r*T);

    // divide by N
    return sum / N;
  }
};

void exmpl_thrust_MC_BS3()
{
  // use 30K independent seeds
  int M = 50000;

  float estimate = thrust::transform_reduce(thrust::counting_iterator<int>(0),
                                            thrust::counting_iterator<int>(M),
                                            estimate_BS3(),
                                            0.0f,
                                            thrust::plus<float>());
  estimate /= M;

  std::cout << std::setprecision(10);
  std::cout << "Option price is approximately " << estimate << std::endl;
  hipDeviceReset();
};