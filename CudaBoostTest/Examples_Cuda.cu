#include "hip/hip_runtime.h"
#include <iostream> 
#include <vector> 
#include <string>
#include <hip/hip_runtime.h> 

#include "Examples_Cuda.h"

#pragma comment(lib, "cudart") 
 
using std::cerr; 
using std::cout; 
using std::endl; 
using std::exception; 
using std::vector; 
 
static const int MaxSize = 96; 
 
// CUDA kernel: cubes each array value 
__global__ void cubeKernel(float* result, float* data) 
{ 
    int idx = threadIdx.x; 
    float f = data[idx]; 
    result[idx] = f * f * f; 
} 
 
// Initializes data on the host 
void InitializeData(vector<float>& data) 
{ 
    for (int i = 0; i < MaxSize; ++i) 
    { 
        data[i] = static_cast<float>(i); 
    } 
} 
 
// Executes CUDA kernel 
void RunCubeKernel(vector<float>& data, vector<float>& result) 
{ 
    const size_t size = MaxSize * sizeof(float); 
 
    // TODO: test for error 
    float* d; 
    float* r; 
    hipError_t hr; 
 
    hr = hipMalloc(reinterpret_cast<void**>(&d), size);            // Could return 46 if device is unavailable. 
    if (hr == cudaErrorDevicesUnavailable) 
    { 
        cerr << "Close all browsers and rerun" << endl; 
        throw std::runtime_error("Close all browsers and rerun"); 
    } 
 
    hr = hipMalloc(reinterpret_cast<void**>(&r), size); 
    if (hr == cudaErrorDevicesUnavailable) 
    { 
        cerr << "Close all browsers and rerun" << endl; 
        throw std::runtime_error("Close all browsers and rerun"); 
    } 
 
    // Copy data to the device 
    hipMemcpy(d, &data[0], size, hipMemcpyHostToDevice); 
 
    // Launch kernel: 1 block, 96 threads 
    // Important: Do not exceed number of threads returned by the device query, 1024 on my computer. 
    cubeKernel<<<1, MaxSize>>>(r, d); 
 
    // Copy back to the host 
    hipMemcpy(&result[0], r, size, hipMemcpyDeviceToHost); 
 
    // Free device memory 
    hipFree(d); 
    hipFree(r); 
}

void exmpl_cuda_cube() {

    std::cout << std::endl << "In this example we show parallelism functionality of Cuda by calculating cubes of numbers from 1 to "<< MaxSize << ":" << std::endl;

    std::cout << "In main." << std::endl; 
 
    // Create sample data 
    std::vector<float> data(MaxSize); 
    InitializeData(data); 
 
    // Compute cube on the device 
    vector<float> cube(MaxSize); 
    RunCubeKernel(data, cube); 
 
    // Print out results 
    cout << "Cube kernel results." << endl << endl; 
 
    for (int i = 0; i < MaxSize; ++i) 
    { 
        cout << cube[i] << " "; 
    }
    cout << endl;
}